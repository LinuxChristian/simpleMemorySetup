
/* ##############################################################
    Copyright (C) 2013 Christian Braedstrup

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
################################################################# */


#include <hip/hip_runtime.h>
#include <stdio.h>
#define real double // Define the precision

// Prototypes
void checkForCudaErrors(const char* checkpoint_description);
void initializeGPU();

int main(int argc, char* argv[])
{

  int xDim = 500; // Node count in x dimension
  int yDim = 500; // Node count in y dimension

  initializeGPU();

  // 
  // Case 1:
  // Linear test
  //

  real *Mat;   // Host pointer
  real *d_Mat; // Device pointer
  Mat = (real*) calloc(xDim, sizeof(real));  // Host memory
  hipMalloc( (void**) &d_Mat, xDim );       // Device memory
  
  printf("Memory copy Host -> Device \n");
  hipMemcpy( d_Mat,   Mat,  xDim, hipMemcpyHostToDevice );
  checkForCudaErrors("Post test 1 memcpy.");

  printf("Clean up \n");
  free( Mat );
  hipFree( d_Mat );

  
  printf("All done");
  return 0;
};


/**
 * This function loads and stores a element from
 * the matrix without shared memory
 * 
 * M_in  Pointer to input matrix
 * M_out Pointer to output matrix
 */
__global__ void cuLoadStoreElement(real *M_in, real *M_out) {

};

/**
 // Check for cuda errors
 // @param checkpoint_description A short message printed to the user
 */
void checkForCudaErrors(const char* checkpoint_description)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("cudaError: %s \n",hipGetErrorString(err));
    exit(EXIT_FAILURE);
  };
}


// Wrapper function for initializing the CUDA components.
// Called from main.cpp
//extern "C"
void initializeGPU()
{
  // Specify target device
  int cudadevice = 0;
  
  // Variables containing device properties
  hipDeviceProp_t prop;
  int devicecount;
  int cudaDriverVersion;
  int cudaRuntimeVersion;
  
  
  // Register number of devices
  hipGetDeviceCount(&devicecount);
  checkForCudaErrors("Initializing GPU!");

  if(devicecount == 0) {
    printf("\nERROR:","No CUDA-enabled devices availible. Bye.\n");
    exit(EXIT_FAILURE);
  } else if (devicecount == 1) {
    printf("\nSystem contains 1 CUDA compatible device.\n","");
  } else {
    printf("\nSystem contains %i CUDA compatible devices.\n",devicecount);
  }
  
    hipGetDeviceProperties(&prop, cudadevice);
    hipDriverGetVersion(&cudaDriverVersion);
    hipRuntimeGetVersion(&cudaRuntimeVersion);
    checkForCudaErrors("Initializing GPU!");

    printf("Using CUDA device ID: %i \n",(cudadevice));
    printf("  - Name: %s, compute capability: %i.%i.\n",prop.name,prop.major,prop.minor);
    printf("  - CUDA Driver version: %i.%i, runtime version %i.%i\n",cudaDriverVersion/1000,cudaDriverVersion%100,cudaRuntimeVersion/1000,cudaRuntimeVersion%100);
    printf("  - Max threads pr. block in x: %i, Max block size in x: %i \n\n",prop.maxThreadsDim[0], prop.maxGridSize[0]);

    // Comment following line when using a system only containing exclusive mode GPUs
    hipChooseDevice(&cudadevice, &prop);
    checkForCudaErrors("Initializing GPU!");
};
