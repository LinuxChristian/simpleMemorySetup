
/* ##############################################################
    Copyright (C) 2013 Christian Braedstrup

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
################################################################# */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h> // CUDA 5.0 Profiler API
#define real double // Define the precision

// Prototypes
void checkForCudaErrors(const char* checkpoint_description);
void initializeGPU();
__global__ void cuLoadStoreElement(real *M_in, real *M_out);

int main(int argc, char* argv[])
{

  int xDim = 500; // Node count in x dimension
  int yDim = 1; // Node count in y dimension
  dim3 BlockSize( 16, 1, 1);
  dim3 GridSize(int(xDim/BlockSize.x), 1, 1);

  
  initializeGPU();

  // 
  // Case 1:
  // Linear test
  //

  real *Mat;      // Host pointer
  real *d_Matin;  // Device pointer to input array
  real *d_Matout; // Device pointer to input array
  Mat = (real*) calloc(xDim, sizeof(real));  // Host memory
  hipMalloc( (void**) &d_Matin , xDim );    // Device memory
  hipMalloc( (void**) &d_Matout, xDim );    // Device memory
  
  printf("Memory copy Host -> Device \n");
  hipMemcpy( d_Matin, Mat,  xDim, hipMemcpyHostToDevice );
  checkForCudaErrors("Test 1 - Memcpy.");

  cuLoadStoreElement<<<BlockSize, GridSize>>>(d_Matin, d_Matout);
  checkForCudaErrors("Test 1 - Kernel call.");

  printf("Clean up \n");
  free( Mat );
  hipFree( d_Matin  );
  hipFree( d_Matout );

  printf("All done");
  return 0;
};


/**
 * This function loads and stores a element from
 * the matrix without shared memory
 * 
 * M_in  Pointer to input matrix
 * M_out Pointer to output matrix
 */
__global__ void cuLoadStoreElement(real *M_in, real *M_out) {

};

/**
 // Check for cuda errors
 // @param checkpoint_description A short message printed to the user
 */
void checkForCudaErrors(const char* checkpoint_description)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("cudaError: %s \n",hipGetErrorString(err));
    exit(EXIT_FAILURE);
  };
}


// Wrapper function for initializing the CUDA components.
// Called from main.cpp
//extern "C"
void initializeGPU()
{
  // Specify target device
  int cudadevice = 0;
  
  // Variables containing device properties
  hipDeviceProp_t prop;
  int devicecount;
  int cudaDriverVersion;
  int cudaRuntimeVersion;
  
  
  // Register number of devices
  hipGetDeviceCount(&devicecount);
  checkForCudaErrors("Initializing GPU!");

  if(devicecount == 0) {
    printf("\nERROR:","No CUDA-enabled devices availible. Bye.\n");
    exit(EXIT_FAILURE);
  } else if (devicecount == 1) {
    printf("\nSystem contains 1 CUDA compatible device.\n","");
  } else {
    printf("\nSystem contains %i CUDA compatible devices.\n",devicecount);
  }
  
    hipGetDeviceProperties(&prop, cudadevice);
    hipDriverGetVersion(&cudaDriverVersion);
    hipRuntimeGetVersion(&cudaRuntimeVersion);
    checkForCudaErrors("Initializing GPU!");
    
    if (cudaRuntimeVersion < 5000) {
      printf("The demo needs CUDA version 5.0 or greater to run!");
      exit(EXIT_FAILURE);
    };

    printf("Using CUDA device ID: %i \n",(cudadevice));
    printf("  - Name: %s, compute capability: %i.%i.\n",prop.name,prop.major,prop.minor);
    printf("  - CUDA Driver version: %i.%i, runtime version %i.%i\n",cudaDriverVersion/1000,cudaDriverVersion%100,cudaRuntimeVersion/1000,cudaRuntimeVersion%100);
    printf("  - Max threads pr. block in x: %i, Max block size in x: %i \n\n",prop.maxThreadsDim[0], prop.maxGridSize[0]);

    // Comment following line when using a system only containing exclusive mode GPUs
    hipChooseDevice(&cudadevice, &prop);
    checkForCudaErrors("Initializing GPU!");
};
